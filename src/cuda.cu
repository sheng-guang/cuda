#include "hip/hip_runtime.h"
#include "cuda.cuh"

#include <cstring>
#include <>
#include "helper.h"

///
/// Algorithm storage
///
// Host copy of input image
Image input_image;
// Host copy of image tiles in each dimension
unsigned int tile_x_count, tile_y_count;
// Pointer to device buffer for calculating the sum of each tile mosaic, this must be passed to a kernel to be used on device
unsigned long long* d_sums;
// Pointer to device buffer for storing the output pixels of each tile, this must be passed to a kernel to be used on device
unsigned char* d_mosaic_value;
// Pointer to device image data buffer, for storing the input image, this must be passed to a kernel to be used on device
unsigned char* d_input_image_data;
// Pointer to device image data buffer, for storing the output image data, this must be passed to a kernel to be used on device
unsigned char* d_output_image_data;
// Pointer to device buffer for the global pixel average sum, this must be passed to a kernel to be used on device
unsigned long long* d_global_pixel_sum;
size_t w_h_c_sizeof_c;
size_t w_h_c;
size_t tx_ty_c;
int channels;
int wide;
//-------------------------------------------------------------------------------------------------------------------
unsigned long long* sums;
unsigned char* cpu_mosaic_value;
Image output_image;
void cuda_begin(const Image *in) {
    // These are suggested CUDA memory allocations that match the CPU implementation
    // If you would prefer, you can rewrite this function (and cuda_end()) to suit your preference

    tile_x_count = in->width / TILE_SIZE;
    tile_y_count = in->height / TILE_SIZE;
    tx_ty_c = tile_x_count * tile_y_count * in->channels;
    // Allocate buffer for calculating the sum of each tile mosaic
    CUDA_CALL(hipMalloc(&d_sums, tx_ty_c * sizeof(unsigned long long)));
    sums = (unsigned long long*)malloc(tx_ty_c * sizeof(unsigned long long));

    // Allocate buffer for storing the output pixel value of each tile
    CUDA_CALL(hipMalloc(&d_mosaic_value, tx_ty_c * sizeof(unsigned char)));
    cpu_mosaic_value = (unsigned char*)malloc(tx_ty_c * sizeof(unsigned char));


    w_h_c_sizeof_c = in->width * in->height * in->channels * sizeof(unsigned char);
    w_h_c = in->width * in->height * in->channels;
    // Allocate copy of input image
    input_image = *in;
    input_image.data = (unsigned char*)malloc(w_h_c_sizeof_c);
    channels= input_image.channels;
    wide = input_image.width;
    memcpy(input_image.data, in->data, w_h_c_sizeof_c);

    // Allocate and fill device buffer for storing input image data
    CUDA_CALL(hipMalloc(&d_input_image_data, w_h_c_sizeof_c));
    CUDA_CALL(hipMemcpy(d_input_image_data, in->data, w_h_c_sizeof_c, hipMemcpyHostToDevice));

    // Allocate device buffer for storing output image data
    CUDA_CALL(hipMalloc(&d_output_image_data, w_h_c_sizeof_c));

    // Allocate and zero buffer for calculation global pixel average
    CUDA_CALL(hipMalloc(&d_global_pixel_sum, in->channels * sizeof(unsigned long long)));
    output_image = *in;
    output_image.data = (unsigned char*)malloc(in->width * in->height * in->channels * sizeof(unsigned char));
}
int cfg1(int total,int cfg2) {
    int re = total / cfg2;
    if (total % cfg2 != 0)re++;
    return re;
}

//1-----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
__global__
void sum(int n, unsigned char* d_input_image_data, unsigned long long* d_sums
    ,int tile_x_count,int channels,int wide) 
{

    unsigned int sum_index= blockDim.x*blockIdx.x+ threadIdx.x;
    if (sum_index >= n)return;

    int t_y = sum_index / (tile_x_count*channels);
    int t_x = (sum_index - t_y * (tile_x_count * channels))/channels;
    int ch = sum_index - t_y * (tile_x_count * channels) - t_x * channels;
    const unsigned int tile_offset = (t_y * tile_x_count * TILE_SIZE * TILE_SIZE + t_x * TILE_SIZE) * channels;

    long long sum = 0;
    int p_y;
    for (p_y = 0; p_y < TILE_SIZE; ++p_y) {
        int p_x;
        for (p_x = 0; p_x < TILE_SIZE; ++p_x) {
            // For each colour channel
            const unsigned int pixel_offset = (p_y * wide + p_x) * channels;
            // Load pixel
            const unsigned char pixel = d_input_image_data[tile_offset + pixel_offset + ch];
            sum += pixel;
        }
    }
    d_sums[sum_index] = sum;
}
void cuda_stage1() {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    //skip_tile_sum(&input_image, sums);
    //printf("sums count: %d\n", tx_ty_c);
    //printf("tile count: %d\n", tx_ty_c/channels);
    //printf("channels count: %d\n",channels);

    int c1 = cfg1(tx_ty_c, 32);
    sum<<<c1,32>>>(tx_ty_c, d_input_image_data, d_sums, tile_x_count, channels,wide);

#ifdef VALIDATION
    hipDeviceSynchronize();
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    CUDA_CALL(hipMemcpy(sums, d_sums, tx_ty_c * sizeof(unsigned long long), hipMemcpyDeviceToHost));
     validate_tile_sum(&input_image, sums);
#endif
}

//2-----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
__global__
void average(int n, unsigned long long* d_sums, unsigned char* d_mosaic_value,int count)
{
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= n)return;
    d_mosaic_value[index] = d_sums[index] / count;
}
void step2_sum_aver(unsigned char* mosaic,int mosaic_count,int channels,unsigned long long* whole_sum,unsigned char*whole_aver)
{
    for (size_t i = 0; i < mosaic_count; i++)
    {
        int ch = i % channels;
        whole_sum[ch] += mosaic[i];
    }
    for (size_t i = 0; i < channels; i++)
    {
        whole_aver[i] = whole_sum[i]/(tile_x_count*tile_y_count);
    }
}
void cuda_stage2(unsigned char* output_global_average) {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    //skip_compact_mosaic(tile_x_count, tile_y_count, sums, cpu_mosaic_value, output_global_average);

    int c1 = cfg1(tx_ty_c, 32);
    average <<<c1, 32 >> > (tx_ty_c, d_sums, d_mosaic_value, TILE_PIXELS);
    hipDeviceSynchronize();
    unsigned long long whole_image_sum[4] = { 0, 0, 0, 0 };
    CUDA_CALL(hipMemcpy(cpu_mosaic_value, d_mosaic_value, tx_ty_c * sizeof(unsigned char), hipMemcpyDeviceToHost));
    step2_sum_aver(cpu_mosaic_value, tx_ty_c, channels, whole_image_sum, output_global_average);
#ifdef VALIDATION
    // TODO: Uncomment and call the validation functions with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    CUDA_CALL(hipMemcpy(cpu_mosaic_value, d_mosaic_value, tx_ty_c * sizeof(unsigned char), hipMemcpyDeviceToHost));
    validate_compact_mosaic(tile_x_count, tile_y_count, sums, cpu_mosaic_value, output_global_average);
#endif    
}


//printf("(%d:tile_x_count)*(%d:t_size_y)*(%d:t_size_x)*(%d:channels)=(%d:count_in_t_line)\n", tile_x_count, t_size_y, t_size_x, channels, count_in_t_line);

//3-----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
__global__
void broadcast(int w_h_c, unsigned char* d_output_image_data, unsigned char* mosaic_value
    ,int tile_x_count, int t_size_y,int t_size_x, int channels,int wide)
{
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= w_h_c)return;
    int count_in_t_line = tile_x_count * t_size_y * t_size_x * channels;
    int t_y = index / count_in_t_line;
    int left = index %( wide*channels);
    int t_x = left / (t_size_x * channels);
    int ch = left % channels;
    d_output_image_data[index] = mosaic_value[(t_y*tile_x_count+t_x)*channels+ch];

}

void cuda_stage3() {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    //skip_broadcast(&input_image, cpu_mosaic_value, &output_image);
    //printf("total : %d\n", w_h_c);
    //printf("channels : %d\n",channels);
    //printf("pix total : %d\n", w_h_c/channels);
    //printf("wide : %d    wide*wide: %d\n", wide,wide*wide);


    int c1 = cfg1(w_h_c, 32);
    broadcast <<<c1, 32 >> > (w_h_c, d_output_image_data, d_mosaic_value
        ,tile_x_count,TILE_SIZE,TILE_SIZE,channels, wide);

#ifdef VALIDATION
    hipDeviceSynchronize();
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    CUDA_CALL(hipMemcpy(output_image.data,d_output_image_data , w_h_c_sizeof_c, hipMemcpyDeviceToHost));
    validate_broadcast(&input_image, cpu_mosaic_value, &output_image);
#endif    
}
void cuda_end(Image *out) {
    // This function matches the provided cuda_begin(), you may change it if desired

    // Store return value
    out->width = input_image.width;
    out->height = input_image.height;
    out->channels = input_image.channels;
    CUDA_CALL(hipMemcpy(out->data, d_output_image_data, out->width * out->height * out->channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
    // Release allocations
    free(input_image.data);
    CUDA_CALL(hipFree(d_mosaic_value));
    CUDA_CALL(hipFree(d_sums));
    CUDA_CALL(hipFree(d_input_image_data));
    CUDA_CALL(hipFree(d_output_image_data));

}
