#include "hip/hip_runtime.h"
#include "cuda.cuh"

#include <cstring>
#include <>
#include "helper.h"

///
/// Algorithm storage
///
// Host copy of input image
Image input_image;
// Host copy of image tiles in each dimension
unsigned int tile_x_count, tile_y_count;
// Pointer to device buffer for calculating the sum of each tile mosaic, this must be passed to a kernel to be used on device
unsigned long long* d_sums;

// Pointer to device buffer for storing the output pixels of each tile, this must be passed to a kernel to be used on device
unsigned char* d_mosaic_value;
// Pointer to device buffer for the global pixel average sum, this must be passed to a kernel to be used on device
unsigned long long* d_global_pixel_sum_origin;
unsigned long long* d_global_pixel_sum_result;


// Pointer to device image data buffer, for storing the input image, this must be passed to a kernel to be used on device
unsigned char* d_input_image_data;
// Pointer to device image data buffer, for storing the output image data, this must be passed to a kernel to be used on device
unsigned char* d_output_image_data;

size_t w_h_c_sizeof_c;
size_t w_h_c;
size_t w_h;
size_t tx_ty_c;
size_t tx_ty;
int channels;
int wide;



//VALIDATION-------------------------------------------------------------------------------------------------------------------
#ifdef VALIDATION
unsigned long long* sums;
unsigned char* cpu_mosaic_value;
Image output_image;
#endif

void cuda_begin(const Image *in) {
    // These are suggested CUDA memory allocations that match the CPU implementation
    // If you would prefer, you can rewrite this function (and cuda_end()) to suit your preference

    tile_x_count = in->width / TILE_SIZE;
    tile_y_count = in->height / TILE_SIZE;
    tx_ty_c = tile_x_count * tile_y_count * in->channels;
    tx_ty = tile_x_count * tile_y_count;
    // Allocate buffer for calculating the sum of each tile mosaic
    CUDA_CALL(hipMalloc(&d_sums, tx_ty_c * sizeof(unsigned long long)));
#ifdef VALIDATION
    sums = (unsigned long long*)malloc(tx_ty_c * sizeof(unsigned long long));
#endif

    // Allocate buffer for storing the output pixel value of each tile
    CUDA_CALL(hipMalloc(&d_mosaic_value, tx_ty_c * sizeof(unsigned char)));
#ifdef VALIDATION
    cpu_mosaic_value = (unsigned char*)malloc(tx_ty_c * sizeof(unsigned char));
#endif


    w_h_c_sizeof_c = in->width * in->height * in->channels * sizeof(unsigned char);
    w_h_c = in->width * in->height * in->channels;
    w_h = in->width * in->height;

    // Allocate copy of input image
    input_image = *in;
    input_image.data = (unsigned char*)malloc(w_h_c_sizeof_c);
    channels= input_image.channels;
    wide = input_image.width;
    memcpy(input_image.data, in->data, w_h_c_sizeof_c);

    // Allocate and fill device buffer for storing input image data
    CUDA_CALL(hipMalloc(&d_input_image_data, w_h_c_sizeof_c));
    CUDA_CALL(hipMemcpy(d_input_image_data, in->data, w_h_c_sizeof_c, hipMemcpyHostToDevice));

    // Allocate device buffer for storing output image data
    CUDA_CALL(hipMalloc(&d_output_image_data, w_h_c_sizeof_c));

    //Allocate and zero buffer for calculation global pixel average
    CUDA_CALL(hipMalloc(&d_global_pixel_sum_origin, tx_ty_c * sizeof(unsigned long long)));
    CUDA_CALL(hipMalloc(&d_global_pixel_sum_result, tx_ty_c * sizeof(unsigned long long)));



#ifdef VALIDATION
    output_image = *in;
    output_image.data = (unsigned char*)malloc(in->width * in->height * in->channels * sizeof(unsigned char));
#endif

}



int cfg1(int total,int cfg2) {
    int re = total / cfg2;
    if (total % cfg2 != 0)re++;
    return re;
}






//1-----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
//template <unsigned int blockSize>
__device__ void warpReduce(volatile unsigned long long* sdata, unsigned int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
    //if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    //if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    //if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    //if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    //if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    //if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
__global__
void sum(int n, unsigned char* d_input_image_data, unsigned long long* d_sums
    ,int tile_x_count,int channels,int wide) 
{
    int t_x = blockIdx.x;
    int t_y = blockIdx.y;
    unsigned int tile_offset = (t_y * tile_x_count * TILE_SIZE * TILE_SIZE + t_x * TILE_SIZE) * channels;


    int p_x = threadIdx.x;
    int p_y = threadIdx.y;
    //printf("%d,%d,%d,%d,\n", t_x, t_y, p_x, p_y);
    unsigned int pixel_offset = (p_y * wide + p_x) * channels;
    int data_index = tile_offset + pixel_offset;

    int offset_x = TILE_SIZE / 2*channels;
    int offset_y = TILE_SIZE / 2 * wide*channels;

    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int blockSize = TILE_PIXELS/4;

    int ch = blockIdx.z;

    __shared__ unsigned long long sdata[TILE_PIXELS/4];
    sdata[tid] = d_input_image_data[data_index + ch] + 
        d_input_image_data[data_index+offset_x + ch] + d_input_image_data[data_index+offset_y + ch] + 
        d_input_image_data[data_index + offset_x +offset_y+ ch];
    __syncthreads();

    //// do reduction in shared mem
    //for (unsigned int s = 1; s < blockSize; s *= 2) {
    //    if (tid % (2 * s) == 0) {
    //        sdata[tid] += sdata[tid + s];
    //    }
    //    __syncthreads();
    //}

    //for (unsigned int s = 1; s < blockSize; s *= 2) {
    //    int index = 2 * s * tid;
    //    if (index < blockSize) {
    //        sdata[index] += sdata[index + s];
    //    }
    //    __syncthreads();
    //}
    for (unsigned int s = blockSize / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid < 32) warpReduce(sdata, tid);
    int T_Index = (t_y * tile_x_count + t_x) * channels;
    // write result for this block to global mem
    if (tid == 0) d_sums[T_Index+ch] = sdata[0];

}


//0.271ms
//4096:6.693ms
//1.271ms
void cuda_stage1() {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    //skip_tile_sum(&input_image, sums);
    //printf("sums count: %d\n", tx_ty_c);
    //printf("tile count: %d\n", tx_ty_c/channels);
    //printf("channels count: %d\n",channels);

    //int c1 = cfg1(tx_ty_c, 32);
    dim3 blocks;
    blocks.x = tile_x_count;
    blocks.y = tile_y_count;
    blocks.z = channels;

    dim3 threads;
    threads.x = TILE_SIZE/2;
    threads.y = TILE_SIZE/2;
    threads.z = 1;
    sum<<<blocks, threads >>>(tx_ty_c, d_input_image_data, d_sums
        , tile_x_count, channels,wide);

#ifdef VALIDATION
    hipDeviceSynchronize();
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    CUDA_CALL(hipMemcpy(sums, d_sums, tx_ty_c * sizeof(unsigned long long), hipMemcpyDeviceToHost));
     validate_tile_sum(&input_image, sums);
#endif
}

//2-----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------


//__global__
//void average(int n, unsigned long long* d_sums, unsigned long long* d_global_pixel_sum, unsigned char* d_mosaic_value,int count,int channels)
//{
//    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
//    if (index >= n)return;
//    int sum_index = index * channels;
//    for (int i = 0; i < channels; i++)
//    {
//        int to = sum_index + i;
//        d_mosaic_value[to] = d_sums[to] / count;
//        d_global_pixel_sum[to] = d_mosaic_value[to];
//    }
//
//}
__global__
void average2(int n, unsigned long long* d_sums, unsigned long long* d_global_pixel_sum, unsigned char* d_mosaic_value, int count)
{
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index >= n)return;
    d_mosaic_value[index] = d_sums[index] / count;
    d_global_pixel_sum[index] = d_mosaic_value[index];
}


//__global__
//void sun_4(int count,unsigned long long* arr, unsigned long long* sum,int _channel)
//{
////#define _channel 3
//    unsigned int thread_index=blockDim.x*blockIdx.x+threadIdx.x;
//    int to = thread_index * _channel;
//    int tid = thread_index *4* _channel;
//    long long re[4];
//
//    //printf("from to  %d  %d  \n", tid, tid + _channel * 3 + 2);
//    
//     for (int i = 0; i < _channel; i++)
//     {
//         re[i] = arr[i+tid] + arr[i+tid + _channel] + arr[i+tid + _channel * 2] + arr[i+tid + _channel * 3];
//         //__syncthreads();
//         //sum[to + i] = re[i];
//     }
//    __syncthreads();
//    for (int i = 0; i < _channel; i++)
//    {
//        sum[to + i] = re[i];
//    }
//
//    //printf("set %d %d %d\n", to, to + 1, to + 2);
//}

__global__
void sun_4_v2(int count, unsigned long long* arr, unsigned long long* sum, int _channel)
{
    //#define _channel 3
    unsigned int thread_index = blockDim.x * blockIdx.x + threadIdx.x;
    int to = thread_index * _channel;
    int tid = thread_index  * _channel*4;

    for (int i = 0; i < _channel; i++)
    {
        sum[to + i] = arr[i + tid] + arr[i + tid + _channel] + arr[i + tid + _channel * 2] + arr[i + tid + _channel * 3];
    }


}

//0.167ms
void cuda_stage2(unsigned char* output_global_average) {
    // Optionally during development call the skip function with the correct inputs to skip this stage
    //skip_compact_mosaic(tile_x_count, tile_y_count, sums, cpu_mosaic_value, output_global_average);
    //{
    //    int c1 = cfg1(tx_ty, 32);
    //    average << <c1, 32 >> > (tx_ty, d_sums, d_global_pixel_sum_result, d_mosaic_value, TILE_PIXELS,channels);
    //} 
    {
        int c1 = cfg1(tx_ty_c, 32);
        average2 << <c1, 32 >> > (tx_ty_c, d_sums, d_global_pixel_sum_result, d_mosaic_value, TILE_PIXELS);
    }
    //{
    //    int count = tx_ty;
    //    //printf("total= %d\n" , tx_ty_c);
    //    while (count>=4)
    //    {
    //        count >>= 2;
    //        //printf("count= %d\n", count);
    //        int c1 = cfg1(count, 32);
    //        sun_4 << <c1, 32 >> > (count,d_global_pixel_sum, d_global_pixel_sum,channels);
    //    }
    //}
    {
        //todo count
        int count = tx_ty;
        while (count >= 4)
        {
            //count/=4;
            count >>= 2;
            int c1 = cfg1(count, 32);
            //exchange
            unsigned long long* temp = d_global_pixel_sum_origin;
            d_global_pixel_sum_origin = d_global_pixel_sum_result;
            d_global_pixel_sum_result = temp;
            //run
            sun_4_v2 << <c1, 32 >> > (count, d_global_pixel_sum_origin, d_global_pixel_sum_result, channels);
        }
    }


    hipDeviceSynchronize();

    unsigned long long arr4[] = { 0,0,0,0 };
    CUDA_CALL(hipMemcpy(arr4, d_global_pixel_sum_result, channels * sizeof(unsigned long long), hipMemcpyDeviceToHost));


    for (size_t i = 0; i < channels; i++)
    {
        output_global_average[i] = arr4[i] / (tile_x_count * tile_y_count);
    }
#ifdef VALIDATION
    // TODO: Uncomment and call the validation functions with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    CUDA_CALL(hipMemcpy(cpu_mosaic_value, d_mosaic_value, tx_ty_c * sizeof(unsigned char), hipMemcpyDeviceToHost));
    validate_compact_mosaic(tile_x_count, tile_y_count, sums, cpu_mosaic_value, output_global_average);
#endif    
}



//3-----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------


//__global__
//void broadcast(int w_h, unsigned char* d_output_image_data, unsigned char* mosaic_value
//    ,  int count_in_tile_line,  int count_in_img_line,  int t_size
//    ,  int tile_x_count, int channels)
//{
//    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
//    if (index >= w_h)return;
//    int t_y = index / count_in_tile_line;
//    int left = index % count_in_img_line;
//    int t_x = left / t_size;
//
//    int index1 = index * channels;
//    int index2 = (t_y * tile_x_count + t_x) * channels;
//
//    for (size_t i = 0; i < channels; i++)
//    {
//        d_output_image_data[index1 +i] = mosaic_value[index2 + i];
//    }
//}


//__global__
//void broadcast2(int mosaic_count, unsigned char* d_output_image_data, unsigned char* d_mosaic_value
//    , int tile_x_count, int channels, int wide)
//{
//
//    unsigned int mosaic_index = blockDim.x * blockIdx.x + threadIdx.x;
//    if (mosaic_index >= mosaic_count)return;
//
//    unsigned int t_y = mosaic_index / (tile_x_count * channels);
//    unsigned int t_x = (mosaic_index - t_y * (tile_x_count * channels)) / channels;
//    unsigned int ch = mosaic_index - t_y * (tile_x_count * channels) - t_x * channels;
//    unsigned int tile_offset = (t_y * tile_x_count * TILE_SIZE * TILE_SIZE + t_x * TILE_SIZE) * channels;
//
//    unsigned int p_y;
//    for (p_y = 0; p_y < TILE_SIZE; ++p_y) {
//        unsigned int p_x;
//        for (p_x = 0; p_x < TILE_SIZE; ++p_x) {
//            // For each colour channel
//            unsigned int pixel_offset = (p_y * wide + p_x) * channels;
//            // Load pixel
//            d_output_image_data[tile_offset + pixel_offset + ch] = d_mosaic_value[mosaic_index];
//        }
//    }
//}
__global__
void broadcast3(unsigned char* d_output_image_data, unsigned char* mosaic_value
    , unsigned int tile_x_count, unsigned int channels, unsigned int wide)
{
    //if (blockIdx.z >= channels)return;
//#define t_x  blockIdx.x
//#define  t_y  blockIdx.y
    unsigned int t_x = blockIdx.x;
    unsigned int t_y = blockIdx.y;
    unsigned int tile_index = (t_y * tile_x_count + t_x) * channels;
    unsigned int tile_offset = (t_y * tile_x_count * TILE_PIXELS + t_x * TILE_SIZE) * channels;

//#define p_x  threadIdx.x
//#define  p_y  threadIdx.y
    unsigned int p_x = threadIdx.x;
    unsigned int p_y = threadIdx.y;
    unsigned int pixel_offset = (p_y * wide + p_x) * channels;
    unsigned int data_index = tile_offset + pixel_offset;

    //#define  ch  blockIdx.z

        //unsigned int ch = blockIdx.z;

    int offset_x = TILE_SIZE / 2 * channels;
    int offset_y = TILE_SIZE / 2 * wide * channels;
    for (int ch = 0; ch < channels; ch++)
    {
        d_output_image_data[data_index + ch] = mosaic_value[tile_index + ch];
        d_output_image_data[data_index + ch + offset_x] = mosaic_value[tile_index + ch];
        d_output_image_data[data_index + ch + offset_y] = mosaic_value[tile_index + ch];
        d_output_image_data[data_index + ch + offset_x + offset_y] = mosaic_value[tile_index + ch];
    }
}

//1.104ms
void cuda_stage3() {
    //{
    //    int c1 = cfg1(w_h, 32);
    //    broadcast << <c1, 32 >> > (w_h, d_output_image_data, d_mosaic_value
    //        , tile_x_count * TILE_PIXELS, wide, TILE_SIZE
    //        , tile_x_count, channels);
    //}

    //{
    //    broadcast2 << <c1, 32 >> > (tx_ty_c, d_output_image_data, d_mosaic_value
    //        , tile_x_count, channels, wide);
    //}


    {
        dim3 blocks_3;
        dim3 threads_3;
        blocks_3.x = tile_x_count;
        blocks_3.y = tile_y_count;
        blocks_3.z = 1;

        threads_3.x = TILE_SIZE / 2;
        threads_3.y = TILE_SIZE / 2;
        threads_3.z = 1;
        broadcast3 << <blocks_3, threads_3 >> > (d_output_image_data, d_mosaic_value,
            tile_x_count, channels, wide);
    }
#ifdef VALIDATION
    // TODO: Uncomment and call the validation function with the correct inputs
    // You will need to copy the data back to host before passing to these functions
    // (Ensure that data copy is carried out within the ifdef VALIDATION so that it doesn't affect your benchmark results!)
    hipDeviceSynchronize();
    CUDA_CALL(hipMemcpy(output_image.data,d_output_image_data , w_h_c_sizeof_c, hipMemcpyDeviceToHost));
    validate_broadcast(&input_image, cpu_mosaic_value, &output_image);
#endif    
}
void cuda_end(Image *out) {
    // This function matches the provided cuda_begin(), you may change it if desired

    // Store return value
    out->width = input_image.width;
    out->height = input_image.height;
    out->channels = input_image.channels;
    CUDA_CALL(hipMemcpy(out->data, d_output_image_data, out->width * out->height * out->channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
    // Release allocations
    free(input_image.data);
    CUDA_CALL(hipFree(d_mosaic_value));
    CUDA_CALL(hipFree(d_sums));
    CUDA_CALL(hipFree(d_input_image_data));
    CUDA_CALL(hipFree(d_output_image_data));
    CUDA_CALL(hipFree(d_global_pixel_sum_origin));
    CUDA_CALL(hipFree(d_global_pixel_sum_result));

#ifdef VALIDATION
    free(sums);
    free(cpu_mosaic_value);
    free(output_image.data);
#endif    

}
